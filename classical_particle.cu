#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <random>

#define DIMS (2)
#define NUM_THREADS (1024)
#define SCALE (0.1)
#define VEL_SCALE (1.0)

// This is supposed to be dynamically compiled with these options
// #define NUM_BLOCKS (5000)
// #define NUM_FRAMES (600)
// #define DT (0.05)
// #define WIDTH (284)
// #define HEIGHT (160)
// #define SHIFT (-5)
// #define PUSH (0.3)
// #define INITIAL_DISTRIBUTION (1)
// #define FORCE_TYPE (1)
// #define EXPOSURE (1)
// #define SHOW_MOMENTUM (0)
// #define MEASUREMENT_TYPE (0)

__device__
void force_barrier(float *pos, float *vel) {
  float x = pos[0];
  float y = pos[1];
  float ax = pow(x, 3) * exp(-pow(x, 4));
  float ay = -1e-4*pow(y, 3);
  vel[0] += ax * DT;
  vel[1] += ay * DT;
}

__device__
void force_double_slit(float *pos, float *vel) {
  float x = pos[0];
  float y = pos[1];
  float wall = exp(-pow(4*(x+1), 4));
  float holes = 1 - exp(-pow(4*(y+1), 4)) - exp(-pow(4*(y-1), 4));
  float ax = 1024 * pow(x+1, 3) * wall * holes;
  float ay = -(pow(y+1, 3) * exp(-pow(4*(y+1), 4)) + pow(y-1, 3) * exp(-pow(4*(y-1), 4))) * wall * 1024;
  vel[0] += ax * DT;
  vel[1] += ay * DT;
}

__device__
void force_mirror(float *pos, float *vel) {
  float x = pos[0];
  float y = pos[1];
  float curve = x - 5.5 + 0.07 * pow(y, 2);
  float potential = exp(-pow(curve, 4));
  float ax = 4 * pow(curve, 3) * potential;
  float ay = 0.07 * 2 * y * ax;
  vel[0] += ax * DT;
  vel[1] += ay * DT;
}

__global__
void step(float *pos, float *vel, int *counts)
{
  int index = DIMS * (threadIdx.x + NUM_THREADS * blockIdx.x);
  pos[index] += vel[index] * DT;
  pos[index + 1] += vel[index + 1] * DT;
  #if (FORCE_TYPE == 1)
    force_barrier(pos+index, vel+index);
  #elif (FORCE_TYPE == 2)
    force_double_slit(pos+index, vel+index);
  #elif (FORCE_TYPE == 3)
    force_mirror(pos + index, vel + index);
  #endif
  #if SHOW_MOMENTUM
    int i = (int)floor(vel[index] * HEIGHT * VEL_SCALE + WIDTH * 0.5);
    int j = (int)floor(vel[index+1] * HEIGHT * VEL_SCALE + HEIGHT * 0.5);
  #else
    int i = (int)floor(pos[index] * HEIGHT * SCALE + WIDTH * 0.5);
    int j = (int)floor(pos[index+1] * HEIGHT * SCALE + HEIGHT * 0.5);
  #endif
  if (i >= 0 && i < WIDTH && j >= 0 && j < HEIGHT) {
    atomicAdd(counts + (i + j * WIDTH), 1);
  }
}

int main(void)
{
  int N = NUM_THREADS * NUM_BLOCKS;
  float *pos, *vel;
  double t = 0;
  #if (MEASUREMENT_TYPE)
    int measurement_done = false;
  #endif

  int *counts;

  std::random_device dev;
  std::mt19937 rng(dev());
  std::normal_distribution<float> dist(0, 1);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&pos, N*DIMS*sizeof(float));
  hipMallocManaged(&vel, N*DIMS*sizeof(float));
  hipMallocManaged(&counts, WIDTH*HEIGHT*sizeof(int));

  #if (INITIAL_DISTRIBUTION == 1)
    for (int i = 0; i < N; i++) {
      pos[2*i] = dist(rng) * 0.5 + SHIFT;
      pos[2*i+1] = dist(rng) * 0.5;
      vel[2*i] = dist(rng) * 0.1 + PUSH;
      vel[2*i+1] = dist(rng) * 0.1;
    }
  #elif (INITIAL_DISTRIBUTION == 2)
    for (int i = 0; 2*i < N; ++i) {
      pos[4*i] = dist(rng) * 0.1 - 3;
      pos[4*i+1] = dist(rng) * 0.1 + 1;
      pos[4*i+2] = dist(rng) * 0.5 + 3;
      pos[4*i+3] = dist(rng) * 0.5 - 0.5;
      vel[4*i] = dist(rng) * 0.1 + PUSH;
      vel[4*i+1] = dist(rng) * 0.1;
      vel[4*i+2] = dist(rng) * 0.1 - PUSH;
      vel[4*i+3] = dist(rng) * 0.1;
    }
  #endif

  for (int i = 0; i < NUM_FRAMES; ++i) {
    for (int j = 0; j < EXPOSURE; ++j) {
      t += DT;
      step<<<N / NUM_THREADS, NUM_THREADS>>>(pos, vel, counts);
    }
    hipDeviceSynchronize();
    #if (MEASUREMENT_TYPE)
      if (t >= 1.0 and !measurement_done) {
        for (int i = 0; i < N; ++i) {
          float x = pos[2*i];
          float y = pos[2*i+1];
          int condition = 0.3 < x && x < 1.8 && -0.1 > y && y > -1.6;
          #if (MEASUREMENT_TYPE == 1)
            if (!condition) {
          #else
            if (condition) {
          #endif
            pos[2*i] = 5;
            pos[2*i+1] = 5;
            vel[2*i] = 0;
            vel[2*i + 1] = 0;
          }
        }
        measurement_done = true;
      }
    #endif
    fwrite(counts, sizeof(int), WIDTH * HEIGHT, stdout);
    for (int i = 0; i < WIDTH * HEIGHT; ++i) {
      counts[i] = 0;
    }
  }

  // Free memory
  hipFree(pos);
  hipFree(vel);
  hipFree(counts);

  return 0;
}
