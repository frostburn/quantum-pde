#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <random>

#define DIMS (2)
#define NUM_THREADS (1024)
#define SCALE (0.1)

// This is supposed to be dynamically compiled with these options
// #define NUM_BLOCKS (5000)
// #define NUM_FRAMES (600)
// #define DT (0.05)
// #define WIDTH (284)
// #define HEIGHT (160)
// #define PUSH (0.3)
// #define INITIAL_DISTRIBUTION (1)
// #define FORCE_TYPE (1)

__device__
void force_barrier(float *pos, float *vel) {
  float x = pos[0];
  float y = pos[1];
  float ax = pow(x, 3) * exp(-pow(x, 4));
  float ay = -1e-4*pow(y, 3);
  vel[0] += ax * DT;
  vel[1] += ay * DT;
}

__device__
void force_double_slit(float *pos, float *vel) {
  float x = pos[0];
  float y = pos[1];
  float wall = exp(-pow(4*(x+1), 4));
  float holes = 1 - exp(-pow(4*(y+1), 4)) - exp(-pow(4*(y-1), 4));
  float ax = 1024 * pow(x+1, 3) * wall * holes;
  float ay = -(pow(y+1, 3) * exp(-pow(4*(y+1), 4)) + pow(y-1, 3) * exp(-pow(4*(y-1), 4))) * wall * holes * 1024;
  vel[0] += ax * DT;
  vel[1] += ay * DT;
}

__global__
void step(float *pos, float *vel, int *counts)
{
  int index = DIMS * (threadIdx.x + NUM_THREADS * blockIdx.x);
  pos[index] += vel[index] * DT;
  pos[index + 1] += vel[index + 1] * DT;
  #if (FORCE_TYPE == 1)
    force_barrier(pos+index, vel+index);
  #elif (FORCE_TYPE == 2)
    force_double_slit(pos+index, vel+index);
  #endif
  int i = (int)floor(pos[index] * HEIGHT * SCALE + WIDTH * 0.5);
  int j = (int)floor(pos[index+1] * HEIGHT * SCALE + HEIGHT * 0.5);
  if (i >= 0 && i < WIDTH && j >= 0 && j < HEIGHT) {
    atomicAdd(counts + (i + j * WIDTH), 1);
  }
}

int main(void)
{
  int N = NUM_THREADS * NUM_BLOCKS;
  float *pos, *vel;

  int *counts;

  std::random_device dev;
  std::mt19937 rng(dev());
  std::normal_distribution<float> dist(0, 1);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&pos, N*DIMS*sizeof(float));
  hipMallocManaged(&vel, N*DIMS*sizeof(float));
  hipMallocManaged(&counts, WIDTH*HEIGHT*sizeof(int));

  for (int i = 0; i < N; i++) {
    #if (INITIAL_DISTRIBUTION == 1)
      pos[2*i] = dist(rng) * 0.5 - 5;
      pos[2*i+1] = dist(rng) * 0.5;
      vel[2*i] = dist(rng) * 0.1 + PUSH;
      vel[2*i+1] = dist(rng) * 0.1;
    #endif
  }

  for (int i = 0; i < NUM_FRAMES; ++i) {
    step<<<N / NUM_THREADS, NUM_THREADS>>>(pos, vel, counts);
    hipDeviceSynchronize();
    fwrite(counts, sizeof(int), WIDTH * HEIGHT, stdout);
    for (int i = 0; i < WIDTH * HEIGHT; ++i) {
      counts[i] = 0;
    }
  }

  // Free memory
  hipFree(pos);
  hipFree(vel);
  hipFree(counts);

  return 0;
}
